#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include <iostream>
#include "stb_image.h"
#include "stb_image_write.h"
#include <stdio.h>

#define maskCols 3
#define maskRows 3
#define imgchannels 1

__constant__ float kernelmatrix[3][3];

using namespace std;

/*void sequentialConvolution(const unsigned char*inputImage,const float * kernel ,unsigned char * outputImageData, int kernelSizeX, int kernelSizeY, int dataSizeX, int dataSizeY, int channels)
{
    int i, j, m, n, mm, nn;
    int kCenterX, kCenterY;                         // center index of kernel
    float sum;                                      // accumulation variable
    int rowIndex, colIndex;                         // indice di riga e di colonna

    const unsigned char * inputImageData = inputImage;
    kCenterX = kernelSizeX / 2;
    kCenterY = kernelSizeY / 2;

    for (int k=0; k<channels; k++) {                    //cycle on channels
        for (i = 0; i < dataSizeY; ++i)                //cycle on image rows
        {
            for (j = 0; j < dataSizeX; ++j)            //cycle on image columns
            {
                sum = 0;
                for (m = 0; m < kernelSizeY; ++m)      //cycle kernel rows
                {
                    mm = kernelSizeY - 1 - m;       // row index of flipped kernel

                    for (n = 0; n < kernelSizeX; ++n)  //cycle on kernel columns
                    {
                        nn = kernelSizeX - 1 - n;   // column index of flipped kernel

                        // indexes used for checking boundary
                        rowIndex = i + m - kCenterY;
                        colIndex = j + n - kCenterX;

                        // ignore pixels which are out of bound
                        if (rowIndex >= 0 && rowIndex < dataSizeY && colIndex >= 0 && colIndex < dataSizeX)
                            sum += inputImageData[(dataSizeX * rowIndex + colIndex)*channels + k] * kernel[kernelSizeX * mm + nn];
                    }
                }
                outputImageData[(dataSizeX * i + j)*channels + k] = sum;

            }
        }
    }
}*/


__global__ void Convolution(unsigned char* image, unsigned char* convolutedoutput, int width, int kernelsize){

    int anchor=kernelsize/2;

    __shared__ char subimg[18][18];

    int tx, ty, bx, by;
    bx=blockIdx.x;
    by=blockIdx.y;
    tx=threadIdx.x;
    ty=threadIdx.y;

    int row=by*blockDim.y + ty;
    int col=bx*blockDim.x + tx;

    if(row<width && col<width){
        subimg[ty + anchor][tx + anchor]=image[row*width + col];
    }
    __syncthreads();


    if(row<anchor){
        subimg[ty][tx]=0;
        subimg[ty][tx+anchor]=0;
    }

    if(col<anchor){
        subimg[ty][tx]=0;
        subimg[ty+anchor][tx]=0;
    }

    if(width-row-1<anchor){
        subimg[ty+2*anchor][tx]=0;
        subimg[ty+2*anchor][tx+anchor]=0;
    }


    if(width-col-1<anchor){
        subimg[ty][tx+2*anchor]=0;
        subimg[ty+anchor][tx+2*anchor]=0;
    }

    if(ty<anchor && by>=1){
        subimg[ty][tx+anchor]=image[(row-anchor)*width+col];
        if(tx<anchor && bx>=1){
            subimg[ty][tx]=image[(row-anchor)*width+col-anchor];
        }
        if((tx+anchor)>(blockDim.x-1) && bx<(width/blockDim.x-1)){
            subimg[ty][tx+(2*anchor)]=image[(row-anchor)*width+col+anchor];
        }
    }

    if(tx<anchor && bx>=1){
        subimg[ty+anchor][tx]=image[row*width+col-anchor];
    }

    if(bx<(width/blockDim.x-1) && (tx+anchor>=(blockDim.x))){
        subimg[ty+anchor][tx+(2*anchor)]=image[row*width+col+anchor];
    }

    if(by<(width/blockDim.y-1) && ((ty+anchor)>=blockDim.y)){
        subimg[ty+(2*anchor)][tx+anchor]=image[(row+anchor)*width+col];
        if(tx<anchor && bx>=1){
            subimg[ty+(2*anchor)][tx]=image[(row+anchor)*width+col-anchor];
        }
        if((tx+anchor)>(blockDim.x-1) && bx<(width/blockDim.x-1)){
            subimg[ty+(2*anchor)][tx+(2*anchor)]=image[(row+anchor)*width+col+anchor];
        }
    }

    __syncthreads();

    int temp=0;

    int r=ty+anchor, c=tx+anchor;
    int kernelrow=0, kernelcol=0;

    for(int i=r-anchor;i<=r+anchor;i++){
        kernelcol=0;
        for(int j=c-anchor;j<=c+anchor;j++){
            temp+=subimg[i][j]*kernelmatrix[kernelrow][kernelcol];
            ++kernelcol;
        }
        ++kernelrow;
    }

    __syncthreads();

    convolutedoutput[row*width+col]=temp;
}

int main(){
    int width, height, bpp, err;
    unsigned char *seq_img, *d_img, *d_output;

    const unsigned char* image = stbi_load( "image64.png", &width, &height, &bpp, imgchannels );
    //img = (unsigned char*)malloc(width*height*sizeof(unsigned char));
    seq_img = (unsigned char*)malloc(width*height*sizeof(unsigned char));

    err=hipMalloc((void**)&d_img, width*height*sizeof(unsigned char));
    err=hipMemcpy(d_img, image, width*height*sizeof(unsigned char), hipMemcpyHostToDevice);

    err=hipMalloc((void**)&d_output, width*height*sizeof(unsigned char));

    cout << "height " << height << " " << width << std::endl; 

    dim3 dimBlock(4,4);
    dim3 dimGrid(16,16);

    float kernel[maskRows][maskCols], kernel1[maskRows][maskCols];
    for(int i=0; i< maskCols; i++){
        for(int j=0;j<maskCols;j++){
            kernel[i][j] = 1.0/(maskRows*maskCols);
            kernel1[i][j] = kernel[i][j];
        }
    }

    for(int i=0; i< maskCols; i++){
        for(int j=0;j<maskCols;j++){
            kernel[i][j] = kernel1[maskRows-i-1][maskCols-j-1];
        }
    }

    err=hipMemcpyToSymbol(HIP_SYMBOL(kernelmatrix), kernel, maskRows*maskCols*sizeof(float));

    Convolution<<<dimBlock, dimGrid>>>(d_img, d_output, width, maskRows);

    err=hipMemcpy(seq_img, d_output, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost);

//sequentialConvolution(image, hostMaskData, seq_img, maskRows, maskCols, width, height, imgchannels);

    stbi_write_png("mynew_seq.png", width, height, imgchannels, seq_img, 0);
    stbi_write_png("mynew_seq1.png", width, height, imgchannels, image, 0);


    /***************************************/

    // Add cuda code here
    return 0;
}